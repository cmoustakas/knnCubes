#include "hip/hip_runtime.h"
/**
	Author: Chares Moustakas
	AEM   : 8860
	Prof  : Nikolaos Pitsianis,Dimitrios Floros
	email : <charesmoustakas@gmail.com> <cmoustakas@ece.auth.gr>
	Descr : The below code implements kNN algorithm  in parallel computing concept via CUDA.
**/


#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#include<stdbool.h>

#define Pi2  6.28
#define e  2.178
#define FLOAT_APPROACH 100000000000000000000000.1
#define bounds 30000

void generateGaussianCurve(float3 *Points,int hostOfELements);
void generateCubeNetwork(float3 *cubeNetwork,int numberOfCubes,float dim);
int serialKNN(float3 host_queryPoints,float3 *host_constantPoints,int hostOfElements);


float cubeLen = 0.25;


__device__ int findNearestFromPrCandidates(float3 querryPoint,int *primaryCandidates,float3 *constantPoints,float3 cube,float cubeLen,int* numOfElements,bool tangentSearch){

	float minDist = powf(querryPoint.x-constantPoints[primaryCandidates[0]].x,2.0);
	minDist += powf(querryPoint.y-constantPoints[primaryCandidates[0]].y,2.0);
	minDist += powf(querryPoint.z-constantPoints[primaryCandidates[0]].z,2.0);
	// DON'T SQRT ... YOU ARE GONNA HAVE A TIME PENALTY

	float currDistance ;
	int counter = 0,returnId;

	while(counter < bounds){
		currDistance =  powf(querryPoint.x-constantPoints[primaryCandidates[counter]].x,2.0);
		currDistance += powf(querryPoint.y-constantPoints[primaryCandidates[counter]].y,2.0);
		currDistance += powf(querryPoint.z-constantPoints[primaryCandidates[counter]].z,2.0);

		if(currDistance < minDist){
			minDist = currDistance ;
			returnId = primaryCandidates[counter];
		}
		counter ++ ;
	}

	/** THERE ARE 6 SURFACES :
	 * 1=> Z = Zcube(+-)cubeLen
	 * 2=> Y = Ycube (+-)cubeLen
	 * 3=> X = Xcube (+-)cubeLen
	 */
	if(!tangentSearch){
		float minSurfDist = fabs((querryPoint.x - cube.x - cubeLen));
		if(minSurfDist>fabs(-querryPoint.x + cube.x - cubeLen)){minSurfDist = fabs(-querryPoint.x + cube.x - cubeLen);}
		if(minSurfDist>fabs(querryPoint.y - cube.y - cubeLen)){ minSurfDist = fabs(querryPoint.y - cube.y - cubeLen);}
		if(minSurfDist>fabs(-querryPoint.y + cube.y - cubeLen)){ minSurfDist = fabs(-querryPoint.y + cube.y - cubeLen);}
		if(minSurfDist>fabs(querryPoint.z - cube.z - cubeLen)){ minSurfDist = fabs(querryPoint.z - cube.z - cubeLen);}
		if(minSurfDist>fabs(-querryPoint.z + cube.z - cubeLen)){ minSurfDist = fabs(-querryPoint.z + cube.z - cubeLen);}

	// Now we got the minimum distance between our point and cube's surfaces
		if(minDist > minSurfDist){returnId = returnId - (int)numOfElements ;}
	}
	//printf("returnId is : %d \n",returnId);
	return returnId ;
}







__device__ int *findPrimaryCandidates(float3 *constantPoints,int* numOfElements,float3 cube,float cubeLen){
	float xCube = cube.x;
	float yCube = cube.y;
	float zCube = cube.z;

	float xConst,yConst,zConst;
	int prim_counter = 0 ;
	int *primaryCandidates = (int*)malloc(bounds*sizeof(int));

	bool inside_cube ;
	for(int i =0;i<(int)numOfElements;i++){
		xConst = constantPoints[i].x ;
		yConst = constantPoints[i].y ;
		zConst = constantPoints[i].z ;

		inside_cube = ((xConst<cubeLen+xCube)&&(xConst>xCube-cubeLen)&&(yConst<yCube+cubeLen)&&(yConst>yCube-cubeLen)&&(zConst<zCube+cubeLen)&&(zConst>zCube-cubeLen));
		if(inside_cube){
			primaryCandidates[prim_counter] = i ;
			prim_counter++;
			
			if(prim_counter>bounds){break;}
		}
	}
	return primaryCandidates;
}






__device__ int searchInTangentCube(int nearestN,float3 querryPoint,float3 *constantPoints,float3 cube,int* numOfElements,float cubeLen){

	printf("IM INSIDE TANGENT CUBE !\n");
	bool tangentSearch = true ;
	int* primaryCandidatesOfTangentCube = findPrimaryCandidates(constantPoints,numOfElements,cube,cubeLen);
	int nearestTangentId = findNearestFromPrCandidates(querryPoint,primaryCandidatesOfTangentCube,constantPoints,cube,cubeLen,numOfElements,tangentSearch);

	float currDistance =  powf(querryPoint.x-constantPoints[nearestN].x,2.0);
	currDistance += powf(querryPoint.y-constantPoints[nearestN].y,2.0);
	currDistance += powf(querryPoint.z-constantPoints[nearestN].z,2.0);

	float newDistance =  powf(querryPoint.x-constantPoints[nearestTangentId].x,2.0);
	newDistance += powf(querryPoint.y-constantPoints[nearestTangentId].y,2.0);
	newDistance += powf(querryPoint.z-constantPoints[nearestTangentId].z,2.0);

	if(newDistance < currDistance){nearestN = nearestTangentId;}
	return nearestN ;

}





__global__  void procKernel(float3 *querryPoints,float3 *constantPoints,int* numOfElements,int *numOfcubes,float3 *cubeNetwork,int*nearestNeig,int*Dim,float*cubeLeng){
	
	
	int unique_id = threadIdx.x + blockIdx.x*blockDim.x;


	//cubeLen = (float)cubeLeng;
	
	int dim = (int)Dim;
	//cubeLen = cubeLen/2;
	bool inside_cube ;
	// Lets check our cube !!
	float xQ = querryPoints[unique_id].x;
	float yQ = querryPoints[unique_id].y;
	float zQ = querryPoints[unique_id].z;

	float xCube,yCube,zCube;
	int my_cubeId;

	for(int i = 0;i < (int)numOfcubes;i++){
		xCube = cubeNetwork[i].x;
		yCube = cubeNetwork[i].y;
		zCube = cubeNetwork[i].z;
		inside_cube = ((xQ<cubeLen+xCube)&&(xQ>xCube-cubeLen)&&(yQ<yCube+cubeLen)&&(yQ>yCube-cubeLen)&&(zQ<zCube+cubeLen)&&(zQ>zCube-cubeLen));
		
		if(inside_cube){
			my_cubeId = i;
			break ;
		}

	}

	int *primaryCandidates  = findPrimaryCandidates(constantPoints,numOfElements,cubeNetwork[my_cubeId],cubeLen);


	int tangentCubesDirections[6] = {1,-1,-dim,dim,dim*dim,-dim*dim};
	bool searchTangent = false ;


	int nearestN = findNearestFromPrCandidates(querryPoints[unique_id],primaryCandidates,constantPoints,cubeNetwork[my_cubeId],cubeLen,numOfElements,searchTangent);
	//printf(" did i found my nearest from pr candidates ? %d \n",unique_id);

	if(nearestN<0){

		nearestN = nearestN + (int)numOfElements;



		for(int i = 0;i<6;i++){


			if((tangentCubesDirections[i] == 1) && (cubeNetwork[my_cubeId].x < 1 - cubeLen)){
				nearestN = searchInTangentCube(nearestN,querryPoints[unique_id],constantPoints,cubeNetwork[my_cubeId + tangentCubesDirections[i]],numOfElements,cubeLen);
			}
			if((tangentCubesDirections[i] == -1)&&(cubeNetwork[my_cubeId].x > cubeLen)){
				nearestN = searchInTangentCube(nearestN,querryPoints[unique_id],constantPoints,cubeNetwork[my_cubeId + tangentCubesDirections[i]],numOfElements,cubeLen);
			}
			if((tangentCubesDirections[i] == -dim)&&(cubeNetwork[my_cubeId].y > cubeLen)){
				nearestN = searchInTangentCube(nearestN,querryPoints[unique_id],constantPoints,cubeNetwork[my_cubeId + tangentCubesDirections[i]],numOfElements,cubeLen);
			}
			if((tangentCubesDirections[i] == dim) && (cubeNetwork[my_cubeId].y < 1 - cubeLen)){
				nearestN = searchInTangentCube(nearestN,querryPoints[unique_id],constantPoints,cubeNetwork[my_cubeId + tangentCubesDirections[i]],numOfElements,cubeLen);
			}
			if((tangentCubesDirections[i] == - dim*dim)&&(cubeNetwork[my_cubeId].z > cubeLen)){
				nearestN = searchInTangentCube(nearestN,querryPoints[unique_id],constantPoints,cubeNetwork[my_cubeId + tangentCubesDirections[i]],numOfElements,cubeLen);
			}
			if((tangentCubesDirections[i] == dim*dim) && (cubeNetwork[my_cubeId].z < 1-cubeLen)){
				nearestN = searchInTangentCube(nearestN,querryPoints[unique_id],constantPoints,cubeNetwork[my_cubeId + tangentCubesDirections[i]],numOfElements,cubeLen);
			}
		}

	}

	nearestNeig[unique_id] = nearestN ;
	if(nearestNeig[unique_id] > (int)numOfElements){printf("boundary error \n");}
}



int main(int argc,char *argv[]){
	if(argc == 3){
	    struct timeval startwtime, endwtime;
	  	double cuda_time;

		int ind = atoi(argv[1]) ;

		int hostOfElements = pow(2,ind);
		// Cubes dimension:
		int host_dim = atoi(argv[2]) ;
		float cubeLeng = cubeLen;

		float* device_cubeLen;
		hipMalloc(&device_cubeLen,sizeof(float));
		hipMemcpy(device_cubeLen,&cubeLeng,sizeof(float),hipMemcpyHostToDevice);
		
		int* device_dim ;
		hipMalloc(&device_dim,sizeof(int));
		hipMemcpy(device_dim,&host_dim,sizeof(int),hipMemcpyHostToDevice);

		// Number of Elements on device
		int* device_numOfElements;
		hipMalloc(&device_numOfElements,sizeof(int));
		hipMemcpy(device_numOfElements,&hostOfElements,sizeof(int),hipMemcpyHostToDevice);

		/** declare  Arrays in host **/

		float3 *host_queryPoints = new float3[hostOfElements];
		float3 *host_constantPoints = new float3[hostOfElements];
		int *host_nN = (int*)malloc(sizeof(int)*hostOfElements);


		generateGaussianCurve(host_queryPoints,hostOfElements);
		generateGaussianCurve(host_constantPoints,hostOfElements);


		float3 *device_queryPoints ;
		float3 *device_constPoints;
		int *device_nN ;

		hipMalloc(&device_queryPoints,sizeof(float3)*hostOfElements);
		hipMalloc(&device_constPoints,sizeof(float3)*hostOfElements);
		hipMalloc(&device_nN,sizeof(int)*hostOfElements);

		hipMemcpy(device_queryPoints,host_queryPoints,sizeof(float3)*hostOfElements,hipMemcpyHostToDevice);
		hipMemcpy(device_constPoints,host_constantPoints,sizeof(float3)*hostOfElements,hipMemcpyHostToDevice);

		int numOfcubes = host_dim*host_dim*host_dim ;

		float3 *host_cubeNetwork = new float3[numOfcubes];

		int *device_numOfcubes;
		hipMalloc(&device_numOfcubes,sizeof(int));


		float3 *device_cubeNetwork;
		hipMalloc(&device_cubeNetwork,sizeof(float3)*numOfcubes);


		
		generateCubeNetwork(host_cubeNetwork,numOfcubes,cubeLeng);

		hipMemcpy(device_cubeNetwork,host_cubeNetwork,sizeof(float3)*numOfcubes,hipMemcpyHostToDevice);
		hipMemcpy(device_numOfcubes,&numOfcubes,sizeof(int),hipMemcpyHostToDevice);

		gettimeofday (&startwtime, NULL);

		procKernel<<<1,hostOfElements>>>(device_queryPoints,device_constPoints,device_numOfElements,device_numOfcubes,device_cubeNetwork,device_nN,device_dim,device_cubeLen);

		hipMemcpy(host_nN,device_nN,sizeof(int)*hostOfElements,hipMemcpyDeviceToHost);
		gettimeofday (&endwtime, NULL);

		  /* get time in seconds */
		cuda_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
				                      + endwtime.tv_sec - startwtime.tv_sec);

		//printf("\n parallel-cuda algorithm completed in : %f seconds. \n",cuda_time);
		printf("%f \n",cuda_time);
		hipFree(device_nN);
		hipFree(device_cubeNetwork);
		hipFree(device_numOfcubes);
		hipFree(device_queryPoints);
		hipFree(device_constPoints);
		hipFree(device_dim);
		hipFree(device_numOfElements);


		time_t t;
		srand((unsigned) time(&t));


	/**	int serial_nN,randomId,success = 0,fails = 0;

		for(int i=0; i<100 ;i++){
			randomId = rand() % ((int)hostOfElements);
			serial_nN = serialKNN(host_queryPoints[randomId],host_constantPoints,hostOfElements);
				printf("\n random element : %d \n",randomId);
				if(serial_nN != host_nN[randomId]){printf("failed for << %d ---- %d >> \n",serial_nN,host_nN[randomId]);fails++;}
				else{printf("SUCCESSSS !!!!!! for << %d ---- %d >> \n",serial_nN,host_nN[randomId]);success++;}
				printf("querry (%f,%f,%f)  \n",host_queryPoints[randomId].x,host_queryPoints[randomId].y,host_queryPoints[randomId].z);
				printf("cuda : (%f,%f,%f) \n",host_constantPoints[host_nN[randomId]].x,host_constantPoints[host_nN[randomId]].y,host_constantPoints[host_nN[randomId]].z);
				printf("serial : (%f,%f,%f) \n\n",host_constantPoints[serial_nN].x,host_constantPoints[serial_nN].y,host_constantPoints[serial_nN].z);

				printf("\n\n cuda nn : %d \n\n",host_nN[randomId]);
		}


		printf("\n\n TOTAL SUCC: %d | TOTAL FAILS : %d \n\n",success,fails);
**/
	}
	else{ printf("ERROR IN ARGUMENTS ****");}
	
	int *serialN = (int*)malloc(sizeof(int)*hostOfElements);
	gettimeofday (&startwtime, NULL);

	for(int j = 0;j<hostOfElements;j++){
		serialN[j] = serialKNN(host_queryPoints[j],host_constantPoints,hostOfElements);
	}

	gettimeofday (&endwtime, NULL);
	double serialTime = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
					                      + endwtime.tv_sec - startwtime.tv_sec);
	printf("serialTime:  %f \n",serialTime);

	return 0;

}

void generateGaussianCurve(float3 *Points,int hostOfElements){
	srand((unsigned int)time(NULL));
	float medianX = 0,medianY=0;
	float stdDeviationX = 0;
	float stdDeviationY = 0;

	for(int i = 0;i<hostOfElements;i++){
		Points[i].x = ((float)rand()/(float)(RAND_MAX));
		Points[i].x = floor(10000*Points[i].x)/10000;

		Points[i].y =  ((float)rand()/(float)(RAND_MAX));
		Points[i].y = floor(10000*Points[i].y)/10000;

		medianX = medianX + Points[i].y;
		medianY = medianY+Points[i].y;
	}

	for(int i = 0;i<hostOfElements;i++){
		stdDeviationX = stdDeviationX + powf((Points[i].x-medianX),2.0);
		stdDeviationY = stdDeviationY + powf((Points[i].y-medianY),2.0);
	}

	stdDeviationX = stdDeviationX/(hostOfElements-1);

	stdDeviationY = stdDeviationY/(hostOfElements-1);

	medianX=medianX/hostOfElements;
	medianY = medianY/hostOfElements;
	float x,y;
	float toPow;
	//Calculate z-axis function-leg :
	for(int i =0;i<hostOfElements;i++){
		x = Points[i].x;
		y = Points[i].y;
		toPow = -(powf(x-medianX,2.0)/2*powf(stdDeviationX,2.0));
		toPow = toPow -(powf(y-medianY,2.0)/2*powf(stdDeviationY,2.0));
		toPow = toPow/FLOAT_APPROACH;

		Points[i].z = powf(e,toPow);
		Points[i].z = (Points[i].z/(Pi2*stdDeviationX*stdDeviationY))*10000000000000000000000000.0 ;
		Points[i].z = floor(10000*Points[i].z)/10000;

		while(Points[i].z >1){Points[i].z = Points[i].z - 0.1;}
	}

}

/** Most efficient way to express cubes: VIA THEIR CENTER-MASS **/
void generateCubeNetwork(float3 *cubeNetwork,int numberOfCubes,float dim){

	int i = 0;
	float x=dim,y=dim,z=dim ;
	while(z<1){
		while(y<1){
			while(x<1){
				cubeNetwork[i].x = x;
				cubeNetwork[i].y = y;
				cubeNetwork[i].z = z;
				x = x + 2*dim ;
				i++ ;

			}
			y = y+2*dim ;
			x = dim;
		}
		z = z+2*dim;
		y = dim;
	}
}








int serialKNN(float3 host_queryPoints,float3 *host_constantPoints,int hostOfElements){

	int valueReturn  ;
	float minDist,currDistance ;


		minDist  =  powf(host_queryPoints.x-host_constantPoints[0].x,2.0);
		minDist += powf(host_queryPoints.y-host_constantPoints[0].y,2.0);
		minDist += powf(host_queryPoints.z-host_constantPoints[0].z,2.0);

		for(int j =0;j<hostOfElements;j++){
			currDistance  =  powf(host_queryPoints.x-host_constantPoints[j].x,2.0);
			currDistance += powf(host_queryPoints.y-host_constantPoints[j].y,2.0);
			currDistance += powf(host_queryPoints.z-host_constantPoints[j].z,2.0);

			if(minDist > currDistance){
				minDist = currDistance ;
				valueReturn = j;
			}

		}




	return valueReturn ;
}
